#include "hip/hip_runtime.h"
#include <arbor/fvm_types.hpp>

#include "cuda_common.hpp"
#include "matrix_common.hpp"

namespace arb {
namespace gpu {

namespace kernels {
/// GPU implementatin of Hines matrix assembly
/// Flat layout
/// For a given time step size dt
///     - use the precomputed alpha and alpha_d values to construct the diagonal
///       and off diagonal of the symmetric Hines matrix.
///     - compute the RHS of the linear system to solve
template <typename T, typename I>
__global__
void assemble_matrix_flat(
        T* d,
        T* rhs,
        const T* invariant_d,
        const T* voltage,
        const T* current,
        const T* conductivity,
        const T* cv_capacitance,
        const T* cv_area,
        const I* cv_to_cell,
        const T* dt_intdom,
        const I* cell_to_intdom,
        unsigned n)
{
    const unsigned tid = threadIdx.x + blockDim.x*blockIdx.x;

    if (tid<n) {
        auto cid = cv_to_cell[tid];
        auto dt = dt_intdom[cell_to_intdom[cid]];

        // Note: dt==0 case is expected only at the end of a mindelay/2
        // integration period, and consequently divergence is unlikely
        // to be a peformance problem.

        if (dt>0) {
            // The 1e-3 is a constant of proportionality required to ensure that the
            // conductance (gi) values have units μS (micro-Siemens).
            // See the model documentation in docs/model for more information.
            T oodt_factor = 1e-3/dt; // [1/μs]
            T area_factor = 1e-3*cv_area[tid]; // [1e-9·m²]

            auto gi = oodt_factor * cv_capacitance[tid] + area_factor*conductivity[tid]; // [μS]
            d[tid] = gi + invariant_d[tid];
            rhs[tid] = gi*voltage[tid] - area_factor*current[tid];
        }
        else {
            d[tid] = 0;
            rhs[tid] = voltage[tid];
        }
    }
}

/// GPU implementatin of Hines matrix assembly
/// Interleaved layout
/// For a given time step size dt
///     - use the precomputed alpha and alpha_d values to construct the diagonal
///       and off diagonal of the symmetric Hines matrix.
///     - compute the RHS of the linear system to solve
template <typename T, typename I, unsigned BlockWidth, unsigned LoadWidth, unsigned Threads>
__global__
void assemble_matrix_interleaved(
        T* d,
        T* rhs,
        const T* invariant_d,
        const T* voltage,
        const T* current,
        const T* conductivity,
        const T* cv_capacitance,
        const T* area,
        const I* sizes,
        const I* starts,
        const I* matrix_to_cell,
        const T* dt_intdom,
        const I* cell_to_intdom,
        unsigned padded_size, unsigned num_mtx)
{
    static_assert(BlockWidth*LoadWidth==Threads,
        "number of threads must equal number of values to process per block");
    __shared__ T buffer_v[Threads];
    __shared__ T buffer_i[Threads];
    __shared__ T buffer_g[Threads];

    const unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
    const unsigned lid = threadIdx.x;

    const unsigned mtx_id   = tid/LoadWidth;
    const unsigned mtx_lane = tid - mtx_id*LoadWidth;

    const unsigned blk_id   = tid/(BlockWidth*LoadWidth);
    const unsigned blk_row  = lid/BlockWidth;
    const unsigned blk_lane = lid - blk_row*BlockWidth;

    const unsigned blk_pos  = LoadWidth*blk_lane + blk_row;

    const bool do_load  = mtx_id<num_mtx;

    unsigned load_pos  = do_load? starts[mtx_id] + mtx_lane     : 0;
    const unsigned end = do_load? starts[mtx_id] + sizes[mtx_id]: 0;
    unsigned store_pos = blk_id*BlockWidth*padded_size + (blk_row*BlockWidth + blk_lane);

    const unsigned max_size = sizes[0];

    T oodt_factor = 0;
    T dt = 0;
    const unsigned permuted_cid = blk_id*BlockWidth + blk_lane;

    if (permuted_cid<num_mtx) {
        auto cid = matrix_to_cell[permuted_cid];
        dt = dt_intdom[cell_to_intdom[cid]];

        // The 1e-3 is a constant of proportionality required to ensure that the
        // conductance (gi) values have units μS (micro-Siemens).
        // See the model documentation in docs/model for more information.

        oodt_factor = dt>0? T(1e-3)/dt: 0;
    }

    for (unsigned j=0u; j<max_size; j+=LoadWidth) {
        if (do_load && load_pos<end) {
            buffer_v[lid] = voltage[load_pos];
            buffer_i[lid] = current[load_pos];
            buffer_g[lid] = conductivity[load_pos];
        }

        __syncthreads();

        if (j+blk_row<padded_size) {
            T area_factor = T(1e-3)*area[store_pos];
            const auto gi = oodt_factor*cv_capacitance[store_pos] + area_factor*buffer_g[blk_pos];

            if (dt>0) {
                d[store_pos]   = (gi + invariant_d[store_pos]);
                rhs[store_pos] = (gi*buffer_v[blk_pos] - area_factor*buffer_i[blk_pos]);
            }
            else {
                d[store_pos]   = 0;
                rhs[store_pos] = buffer_v[blk_pos];
            }
        }

        __syncthreads();

        store_pos += LoadWidth*BlockWidth;
        load_pos  += LoadWidth;
    }
}

} // namespace kernels

void assemble_matrix_flat(
        fvm_value_type* d,
        fvm_value_type* rhs,
        const fvm_value_type* invariant_d,
        const fvm_value_type* voltage,
        const fvm_value_type* current,
        const fvm_value_type* conductivity,
        const fvm_value_type* cv_capacitance,
        const fvm_value_type* area,
        const fvm_index_type* cv_to_cell,
        const fvm_value_type* dt_intdom,
        const fvm_index_type* cell_to_intdom,
        unsigned n)
{
    constexpr unsigned block_dim = 128;
    const unsigned grid_dim = impl::block_count(n, block_dim);

    kernels::assemble_matrix_flat
        <fvm_value_type, fvm_index_type>
        <<<grid_dim, block_dim>>>
        (d, rhs, invariant_d, voltage, current, conductivity, cv_capacitance,
         area, cv_to_cell, dt_intdom, cell_to_intdom, n);
}

//template <typename T, typename I, unsigned BlockWidth, unsigned LoadWidth, unsigned Threads>
void assemble_matrix_interleaved(
    fvm_value_type* d,
    fvm_value_type* rhs,
    const fvm_value_type* invariant_d,
    const fvm_value_type* voltage,
    const fvm_value_type* current,
    const fvm_value_type* conductivity,
    const fvm_value_type* cv_capacitance,
    const fvm_value_type* area,
    const fvm_index_type* sizes,
    const fvm_index_type* starts,
    const fvm_index_type* matrix_to_cell,
    const fvm_value_type* dt_intdom,
    const fvm_index_type* cell_to_intdom,
    unsigned padded_size, unsigned num_mtx)
{
    constexpr unsigned bd = impl::matrices_per_block();
    constexpr unsigned lw = impl::load_width();
    constexpr unsigned block_dim = bd*lw;

    // The number of threads is threads_per_matrix*num_mtx
    const unsigned grid_dim = impl::block_count(num_mtx*lw, block_dim);

    kernels::assemble_matrix_interleaved
        <fvm_value_type, fvm_index_type, bd, lw, block_dim>
        <<<grid_dim, block_dim>>>
        (d, rhs, invariant_d, voltage, current, conductivity, cv_capacitance, area,
         sizes, starts, matrix_to_cell,
         dt_intdom, cell_to_intdom, padded_size, num_mtx);
}

} // namespace gpu
} // namespace arb
