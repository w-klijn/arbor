#include "hip/hip_runtime.h"
#include "../gtest.h"

#include <vector>

#include <backends/gpu/reduce_by_key.hpp>
#include <memory/memory.hpp>
#include <util/span.hpp>
#include <util/rangeutil.hpp>

using namespace arb;

template <typename T, typename I>
__global__
void reduce_kernel(const T* src, T* dst, const I* index, int n) {
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

    unsigned mask = __ballot_sync(0xffffffff, tid<n);
    if (tid<n) {
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
    }
}

template <typename T>
std::vector<T> reduce(const std::vector<T>& in, size_t n_out, const std::vector<int>& index, unsigned block_dim=128) {
    EXPECT_EQ(in.size(), index.size());
    EXPECT_TRUE(std::is_sorted(index.begin(), index.end()));

    using array = memory::device_vector<T>;
    using iarray = memory::device_vector<int>;

    int n = in.size();

    array  src = memory::make_const_view(in);
    iarray idx = memory::make_const_view(index);
    array  dst(n_out, 0);

    unsigned grid_dim = (n-1)/block_dim + 1;
    reduce_kernel<<<grid_dim, block_dim>>>(src.data(), dst.data(), idx.data(), n);

    std::vector<T> out(n_out);
    memory::copy(dst, memory::make_view(out));

    return out;
}

TEST(reduce_by_key, no_repetitions)
{
    int n = 64;
    {
        std::vector<float> in(n, 1);
        std::vector<int> index = util::assign_from(util::make_span(0, n));

        auto out = reduce(in, n, index);
        for (auto o: out) EXPECT_EQ(o, 1.0f);
    }
    {
        std::vector<double> in(n, 1);
        std::vector<int> index = util::assign_from(util::make_span(0, n));

        auto out = reduce(in, n, index);
        for (auto o: out) EXPECT_EQ(o, 1.0);
    }
}

TEST(reduce_by_key, single_repeated_index)
{
    // Perform reduction of a sequence of 1s of length n
    // The expected result is n
    for (auto n: {1, 2, 7, 31, 32, 33, 63, 64, 65, 128}) {
        std::vector<double> in(n, 1);
        std::vector<int> index(n, 0);

        auto out = reduce(in, 1, index, 32);
        EXPECT_EQ(double(n), out[0]);
    }
    // Perform reduction of an ascending sequence of {1,2,3,...,n}
    // The expected result is n*(n+1)/2
    for (auto n: {1, 2, 7, 31, 32, 33, 63, 64, 65, 128}) {
        std::vector<double> in = util::assign_from(util::make_span(1, n+1));
        std::vector<int> index(n, 0);

        auto out = reduce(in, 1, index);
        EXPECT_EQ(out[0], double((n+1)*n/2));
    }
}

TEST(reduce_by_key, scatter)
{
    std::vector<int> index = {0,0,0,1,2,2,2,2,3,3,7,7,7,7,7,11};
    unsigned n = util::max_value(index)+1;
    std::vector<double> in(index.size(), 1);
    std::vector<double> expected = {3., 1., 4., 2., 0., 0., 0., 5., 0., 0., 0., 1.};

    unsigned m = index.size();

    EXPECT_EQ(n, expected.size());

    auto out = reduce(in, n, index);
    EXPECT_EQ(expected, out);

    // rerun with 7 threads per thread block, to test
    //  * using more than one thread block
    //  * thread blocks that are not a multiple of 32
    //  * thread blocks that are less than 32

    out = reduce(in, n, index, 7);
    EXPECT_EQ(expected, out);
}

// Test kernels that perform more than one reduction in a single invokation.
// Used to reproduce and test for synchronization issues on V100 GPUs.

template <typename T, typename I>
__global__
void reduce_twice_kernel(const T* src, T* dst, const I* index, int n) {
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

    unsigned mask = __ballot_sync(0xffffffff, tid<n);
    if (tid<n) {
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
        gpu::reduce_by_key(src[tid], dst, index[tid], mask);
    }
}

template <typename T>
std::vector<T> reduce_twice(const std::vector<T>& in, size_t n_out, const std::vector<int>& index, unsigned block_dim=128) {
    EXPECT_EQ(in.size(), index.size());
    EXPECT_TRUE(std::is_sorted(index.begin(), index.end()));

    using array = memory::device_vector<T>;
    using iarray = memory::device_vector<int>;

    int n = in.size();

    array  src = memory::make_const_view(in);
    iarray idx = memory::make_const_view(index);
    array  dst(n_out, 0);

    unsigned grid_dim = (n-1)/block_dim + 1;
    reduce_twice_kernel<<<grid_dim, block_dim>>>(src.data(), dst.data(), idx.data(), n);

    std::vector<T> out(n_out);
    memory::copy(dst, memory::make_view(out));

    return out;
}

TEST(reduce_by_key, scatter_twice)
{
    std::vector<int> index = {0,0,0,1,2,2,3,7,7,7,11};
    unsigned n = util::max_value(index)+1;
    std::vector<double> in(index.size(), 1);
    std::vector<double> expected = {6., 2., 4., 2., 0., 0., 0., 6., 0., 0., 0., 2.};

    unsigned m = index.size();

    EXPECT_EQ(n, expected.size());

    auto out = reduce_twice(in, n, index);
    EXPECT_EQ(expected, out);

    // rerun with 7 threads per thread block, to test
    //  * using more than one thread block
    //  * thread blocks that are not a multiple of 32
    //  * thread blocks that are less than 32

    out = reduce_twice(in, n, index, 7);
    EXPECT_EQ(expected, out);
}
