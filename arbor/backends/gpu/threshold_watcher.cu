#include "hip/hip_runtime.h"
#include <cmath>

#include <arbor/fvm_types.hpp>

#include "backends/threshold_crossing.hpp"
#include "cuda_common.hpp"
#include "stack_cu.hpp"

namespace arb {
namespace gpu {

namespace kernel {

template <typename T>
__device__
inline T lerp(T a, T b, T u) {
    return std::fma(u, b, std::fma(-u, a, a));
}

/// kernel used to test for threshold crossing test code.
/// params:
///     t       : current time (ms)
///     t_prev  : time of last test (ms)
///     size    : number of values to test
///     is_crossed  : crossing state at time t_prev (true or false)
///     prev_values : values at sample points (see index) sampled at t_prev
///     index      : index with locations in values to test for crossing
///     values     : values at t_prev
///     thresholds : threshold values to watch for crossings
__global__
void test_thresholds_impl(
    int size,
    const fvm_index_type* cv_to_intdom, const fvm_value_type* t_after, const fvm_value_type* t_before,
    stack_storage<threshold_crossing>& stack,
    fvm_index_type* is_crossed, fvm_value_type* prev_values,
    const fvm_index_type* cv_index, const fvm_value_type* values, const fvm_value_type* thresholds)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    bool crossed = false;
    float crossing_time;

    if (i<size) {
        // Test for threshold crossing
        const auto cv     = cv_index[i];
        const auto cell   = cv_to_intdom[cv];
        const auto v_prev = prev_values[i];
        const auto v      = values[cv];
        const auto thresh = thresholds[i];

        if (!is_crossed[i]) {
            if (v>=thresh) {
                // The threshold has been passed, so estimate the time using
                // linear interpolation
                auto pos = (thresh - v_prev)/(v - v_prev);
                crossing_time = lerp(t_before[cell], t_after[cell], pos);

                is_crossed[i] = 1;
                crossed = true;
            }
        }
        else if (v<thresh) {
            is_crossed[i]=0;
        }

        prev_values[i] = v;
    }

    if (crossed) {
        push_back(stack, {fvm_size_type(i), crossing_time});
    }
}

__global__
extern void reset_crossed_impl(
    int size, fvm_index_type* is_crossed,
    const fvm_index_type* cv_index, const fvm_value_type* values, const fvm_value_type* thresholds)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i<size) {
        is_crossed[i] = values[cv_index[i]] >= thresholds[i];
    }
}

} // namespace kernel

void test_thresholds_impl(
    int size,
    const fvm_index_type* cv_to_intdom, const fvm_value_type* t_after, const fvm_value_type* t_before,
    stack_storage<threshold_crossing>& stack,
    fvm_index_type* is_crossed, fvm_value_type* prev_values,
    const fvm_index_type* cv_index, const fvm_value_type* values, const fvm_value_type* thresholds)
{
    if (size>0) {
        constexpr int block_dim = 128;
        const int grid_dim = impl::block_count(size, block_dim);
        kernel::test_thresholds_impl<<<grid_dim, block_dim>>>(
            size, cv_to_intdom, t_after, t_before, stack, is_crossed, prev_values, cv_index, values, thresholds);
    }
}

void reset_crossed_impl(
    int size, fvm_index_type* is_crossed,
    const fvm_index_type* cv_index, const fvm_value_type* values, const fvm_value_type* thresholds)
{
    if (size>0) {
        constexpr int block_dim = 128;
        const int grid_dim = impl::block_count(size, block_dim);
        kernel::reset_crossed_impl<<<grid_dim, block_dim>>>(size, is_crossed, cv_index, values, thresholds);
    }
}

} // namespace gpu
} // namespace arb
